#include "hip/hip_runtime.h"
#include "global_cuda.cuh"

float oldrand[DIM];
__device__ float oldrand_dev[DIM];
__device__ int jrand_dev;

__global__ void advance_rndgen_kernel () {
    
    unsigned int tid = blockDim.x*blockIdx.x + threadIdx.x;
	
	float new_random;

    if (tid<24) {
        new_random = oldrand_dev[tid]-oldrand_dev[tid+31];
        if (new_random<0.0f)
            new_random = new_random+1.0f;
        oldrand_dev[tid] = new_random;
    }
	__syncthreads();

    if (tid>23 && tid<48) {
        new_random = oldrand_dev[tid]-oldrand_dev[tid-24];
        if (new_random<0.0f)
            new_random = new_random+1.0f;
        oldrand_dev[tid] = new_random;
    }
	__syncthreads();

	if (tid>47 && tid<DIM) {
        new_random = oldrand_dev[tid]-oldrand_dev[tid-24];
        if (new_random<0.0f)
            new_random = new_random+1.0f;
        oldrand_dev[tid] = new_random;
    }
}
void initialize_rndgen_gpu (float seed) {
	
	memset(oldrand,0,DIM*sizeof(float));
	oldrand[DIM-1] = seed;

	float new_random = 0.000000001f;
	float prev_random = seed;
	
	int i; 
	for (int j=1; j<=DIM-1; j++) {
		i = (21*j)%(DIM-1);
		oldrand[i] = new_random;
		new_random = prev_random-new_random;
		if (new_random<0.0f)
			new_random += 1.0f;
		prev_random = oldrand[i];
	}

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(oldrand_dev), oldrand, DIM*sizeof(float), 0, hipMemcpyHostToDevice));

	int jrand = 0;
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(jrand_dev), &jrand, sizeof(int), 0, hipMemcpyHostToDevice));

	dim3 threadsPerBlock(DIM,1,1);
	dim3 blocksPerGrid(1,1,1);

	advance_rndgen_kernel<<<blocksPerGrid, threadsPerBlock>>> ();
    advance_rndgen_kernel<<<blocksPerGrid, threadsPerBlock>>> ();
    advance_rndgen_kernel<<<blocksPerGrid, threadsPerBlock>>> ();
}

// Extract numel float random numbers between 0.0 and 1.0 -including the bounds-
__global__ void extract_rndvec_kernel (float *vout, int numel) {
    
    unsigned int tid = blockDim.x*blockIdx.x + threadIdx.x;
	
	float new_random;

    int nbatch = (numel+jrand_dev-1) / (DIM-1);
	int nrem = (numel+jrand_dev-1) % (DIM-1);
		
	if (tid>=jrand_dev && tid<jrand_dev+numel && tid<DIM)
		vout[tid-jrand_dev] = oldrand_dev[tid];
	
	if (nbatch==0) {
		jrand_dev += numel;
		return;
	} 
		
	for (int i=0; i<nbatch; i++) {

		if (tid<24) {
		new_random = oldrand_dev[tid]-oldrand_dev[tid+31];
		if (new_random<0.0f)
			new_random = new_random+1.0f;
		oldrand_dev[tid] = new_random;
		}
		__syncthreads();
		
		if (tid>23 && tid<48) {
			new_random = oldrand_dev[tid]-oldrand_dev[tid-24];
			if (new_random<0.0f)
				new_random = new_random+1.0f;
			oldrand_dev[tid] = new_random;
		}
		__syncthreads();
		
		if (tid>47 && tid<DIM) {
			new_random = oldrand_dev[tid]-oldrand_dev[tid-24];
			if (new_random<0.0f)
				new_random = new_random+1.0f;
			oldrand_dev[tid] = new_random;
		}
		__syncthreads();

		if (i<nbatch-1) {
			if (tid>=1 && tid<DIM)
				vout[(DIM-jrand_dev) + i*(DIM-1) + (tid-1)] = oldrand_dev[tid];
		} else {
			if (tid>=1 && tid<=nrem)
				vout[(DIM-jrand_dev) + i*(DIM-1) + (tid-1)] = oldrand_dev[tid];
			jrand_dev = nrem + 1;
		}
	}
}
// Extract numel int random numbers between low and high -including the bounds-
__global__ void extract_rndvec_kernel_lowhigh_32s (float *vout, int numel, int low, int high) {
    
    unsigned int tid = blockDim.x*blockIdx.x + threadIdx.x;
	
	float new_random;
	int vtmp;

    int nbatch = (numel+jrand_dev-1) / (DIM-1);
	int nrem = (numel+jrand_dev-1) % (DIM-1);
		
	if (tid>=jrand_dev && tid<jrand_dev+numel && tid<DIM) {
		vtmp = low + (oldrand_dev[tid]*(high-low+1.0f));
		if (vtmp>high) 
			vout[tid-jrand_dev] = (float)high;
		else vout[tid-jrand_dev] = (float)vtmp;
	}
	
	if (nbatch==0) {
		jrand_dev += numel;
		return;
	} 
		
	for (int i=0; i<nbatch; i++) {

		if (tid<24) {
		new_random = oldrand_dev[tid]-oldrand_dev[tid+31];
		if (new_random<0.0f)
			new_random = new_random+1.0f;
		oldrand_dev[tid] = new_random;
		}
		__syncthreads();
		
		if (tid>23 && tid<48) {
			new_random = oldrand_dev[tid]-oldrand_dev[tid-24];
			if (new_random<0.0f)
				new_random = new_random+1.0f;
			oldrand_dev[tid] = new_random;
		}
		__syncthreads();
		
		if (tid>47 && tid<DIM) {
			new_random = oldrand_dev[tid]-oldrand_dev[tid-24];
			if (new_random<0.0f)
				new_random = new_random+1.0f;
			oldrand_dev[tid] = new_random;
		}
		__syncthreads();

		if (i<nbatch-1) {
			if (tid>=1 && tid<DIM) {
				vtmp = low + (oldrand_dev[tid]*(high-low+1.0f));
				if (vtmp>high) 
					vout[(DIM-jrand_dev) + i*(DIM-1) + (tid-1)] = high;
				else vout[(DIM-jrand_dev) + i*(DIM-1) + (tid-1)] = vtmp;
			}
		} else {
			if (tid>=1 && tid<=nrem) {				
				vtmp = low + (oldrand_dev[tid]*(high-low+1.0f));
				if (vtmp>high) 
					vout[(DIM-jrand_dev) + i*(DIM-1) + (tid-1)] = (float)high;
				else vout[(DIM-jrand_dev) + i*(DIM-1) + (tid-1)] = (float)vtmp;
			}
			jrand_dev = nrem + 1;
		}
	}
}
// Extract numel float random numbers between low and high -including the bounds-
__global__ void extract_rndvec_kernel_lowhigh_32f (float *vout, int numel, float low, float high) {
    
    unsigned int tid = blockDim.x*blockIdx.x + threadIdx.x;
	
	float new_random;

    int nbatch = (numel+jrand_dev-1) / (DIM-1);
	int nrem = (numel+jrand_dev-1) % (DIM-1);
		
	if (tid>=jrand_dev && tid<jrand_dev+numel && tid<DIM)
		vout[tid-jrand_dev] = low + oldrand_dev[tid]*(high-low);
	
	if (nbatch==0) {
		jrand_dev += numel;
		return;
	} 
		
	for (int i=0; i<nbatch; i++) {

		if (tid<24) {
		new_random = oldrand_dev[tid]-oldrand_dev[tid+31];
		if (new_random<0.0f)
			new_random = new_random+1.0f;
		oldrand_dev[tid] = new_random;
		}
		__syncthreads();
		
		if (tid>23 && tid<48) {
			new_random = oldrand_dev[tid]-oldrand_dev[tid-24];
			if (new_random<0.0f)
				new_random = new_random+1.0f;
			oldrand_dev[tid] = new_random;
		}
		__syncthreads();
		
		if (tid>47 && tid<DIM) {
			new_random = oldrand_dev[tid]-oldrand_dev[tid-24];
			if (new_random<0.0f)
				new_random = new_random+1.0f;
			oldrand_dev[tid] = new_random;
		}
		__syncthreads();

		if (i<nbatch-1) {
			if (tid>=1 && tid<DIM)
				vout[(DIM-jrand_dev) + i*(DIM-1) + (tid-1)] = low + oldrand_dev[tid]*(high-low);
		} else {
			if (tid>=1 && tid<=nrem)			
				vout[(DIM-jrand_dev) + i*(DIM-1) + (tid-1)] = low + oldrand_dev[tid]*(high-low);
			jrand_dev = nrem + 1;
		}
	}
}

