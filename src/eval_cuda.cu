#include "hip/hip_runtime.h"
#include "global_cuda.cuh"

void evaluate_gpu (population_gpu *pop, int size) {
 
	#ifdef sch1
	
	checkCudaErrors(nppsMul_32f(pop->xreal, pop->xreal, pop->obj, size));
		
	checkCudaErrors(nppsSubC_32f(pop->xreal, 2.0f, pop->obj + size, size));	
	checkCudaErrors(nppsMul_32f_I(pop->obj + size, pop->obj + size, size));

	checkCudaErrors(nppsSet_32f(0, pop->constr_violation, size));

	if (ncon!=0) {
		checkCudaErrors(nppsThreshold_GT_32f(pop->constr, tmp_locs_32f_dev, ncon*size, 0));
		
		// se i vincoli sono pochi, okay cos�
		// altrimenti si potrebbe chiamare hipblasSgemv facendo: transpose(tmp_locs_32f_dev)*ones(ncon,1) -dove ones � gi� disponibile-
		for (int i=0; i<ncon; i++)
			checkCudaErrors(nppsAdd_32f_I(tmp_locs_32f_dev + i*size, pop->constr_violation, size));
	}

	#endif
			
}