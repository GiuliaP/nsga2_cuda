#include "hip/hip_runtime.h"
#include "global_cuda.cuh"

void initialize_pop_gpu (population_gpu *pop, int size) {
	
	if (nreal!=0) {
		
		/*
		int nbatch = (int)floorf((float)(nreal*size)/(DIM-1));
		int nrem = (nreal*size)%(DIM-1);

		dim3 threadsPerBlock(DIM,1,1);
		dim3 blocksPerGrid(1,1,1);
	
		for (int i=0; i<nbatch; i++)
			advance_random_device_copyvector<<<blocksPerGrid, threadsPerBlock>>>(pop->xreal + i*(DIM-1), DIM-1);

		if (nrem!=0)
			advance_random_device_copyvector<<<blocksPerGrid, threadsPerBlock>>>(pop->xreal + nbatch*(DIM-1), nrem);
		*/

		dim3 threadsPerBlock(DIM,1,1);
		dim3 blocksPerGrid(1,1,1);
		
		for (int i=0; i<nreal; i++) 
			extract_rndvec_kernel_lowhigh_32f<<<blocksPerGrid, threadsPerBlock>>>(pop->xreal, nreal*size, min_realvar[i], max_realvar[i]);

		/*
		for (int i=0; i<nreal; i++) 
			checkCudaErrors(nppsMulC_32f_I(max_realvar[i]-min_realvar[i],pop->xreal + i*size,size));
		for (int i=0; i<nreal; i++) 
			checkCudaErrors(nppsAddC_32f_I(min_realvar[i],pop->xreal + i*size,size));
		*/
	}
}