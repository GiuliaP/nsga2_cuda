#include "hip/hip_runtime.h"
#include "global_cuda.cuh"

/* Routine for usual non-domination checking
   It will return the following values
   1 if p dominates q
   -1 if q dominates p
   0 if both p and q are non-dominated */

__device__ void check_dominance_device_masked (float *obj, float *constr_violation, const int size, const int nobj, unsigned p, unsigned q, float *S_loc, char *flag_loc) {

    unsigned char pmeno = (constr_violation[p]<0); 
	unsigned char qmeno = (constr_violation[q]<0);
	unsigned char pzero = (constr_violation[p]==0); 
	unsigned char qzero = (constr_violation[q]==0);

	char mask6 = pzero*qmeno;
	char mask8 = -(pmeno*qzero);
	char mask9 = pmeno*qmeno;
	char mask754321 = 1 - (mask6 - mask8 + mask9);

	float diff = constr_violation[p] - constr_violation[q];
	mask9 *= (char)(diff/fabs(diff));;

	unsigned char flag1 = 0;
    unsigned char flag2 = 0;
	for (int i=0; i<nobj; i++) {
		flag1 = flag1 || (obj[i*size + p] < obj[i*size + q]);
		flag2 = flag2 || (obj[i*size + p] > obj[i*size + q]);
	}
	mask754321 *= (char)(flag1 - flag2);

	// flag_loc contiene la dominance (-1,0,1)
	char mask_tot = mask6 + mask8 + mask9 + mask754321;
	*flag_loc = mask_tot;

	// diversamente dall'algoritmo originale!!!!
	*S_loc = (float)(mask_tot<0);
}

__global__ void check_dominance_kernel_masked (float *obj, float *constr_violation, const int size, const int nobj, float *S_loc, char *flag_loc) {

	unsigned int p = threadIdx.x + blockDim.x*(blockIdx.x/size);
	unsigned int q = blockIdx.x%size;
	unsigned idx = p + size*q;

	check_dominance_device_masked (obj, constr_violation, size, nobj, p, q, S_loc + idx, flag_loc + idx);

	//check_dominance_device_masked (obj, constr_violation, size, nobj, threadIdx.x, blockIdx.x, S_loc + threadIdx.x + blockIdx.x*blockDim.x, flag_loc + threadIdx.x + blockIdx.x*blockDim.x);
}