#include "hip/hip_runtime.h"
#include <thrust/iterator/counting_iterator.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/sort.h>

#include "global_cuda.cuh"

using namespace thrust;
using namespace thrust::placeholders;

template<typename T>
struct is_one : thrust::unary_function<T, bool>
{
    __host__ __device__
    bool operator()(const T &x)
    {
        return (x==1);
    }
};

template<typename T>
struct is_zero : thrust::unary_function<T, bool>
{
    __host__ __device__
    bool operator()(const T &x)
    {
        return (x==0);
    }
};

__global__ void calc_next_front (float *n, const float *S, const int pop_size, int *front_elements_loc, int *front_size_loc, int *front_offset_loc, const int front) {

	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	
	int index;
	float S_loc;
	float n_loc = n[tid];
	int new_front_loc=0;

	for (int k=0; k<front_size_loc[front-1]; k++) {
		S_loc = S[front_elements_loc[front_offset_loc[front-1] + k]*pop_size + tid];
		n_loc -= S_loc;
		new_front_loc = new_front_loc || ( (n_loc==0) && (S_loc==1) );
	}

	if (new_front_loc) {
		index = atomicAdd(front_size_loc + front, 1);
		front_elements_loc[front_offset_loc[front] + index] = tid;
	}

	n[tid] = n_loc;
}

void qsort_front_obj_gpu (int *values, float *keys, int left, int right) {
    
	int index;
    int temp;
    int i, j;
    float pivot;
    
	if (left<right) {
        
		index = rnd (left, right);
        temp = values[right];
        values[right] = values[index];
        values[index] = temp;

        pivot = keys[values[right]];
		i = left-1;
        for (j=left; j<right; j++)
			if (keys[values[j]] <= pivot) {
                i++;
                temp = values[j];
                values[j] = values[i];
                values[i] = temp;
            }
       
        index=i+1;
        temp = values[index];
		values[index] = values[right];
		values[right] = temp;	
       
        qsort_front_obj_gpu (values, keys, left, index-1);
        qsort_front_obj_gpu (values, keys, index+1, right);
    }
}

void assign_crowd_dist_gpu (population_cpu *pop, int size, int front) {
   
	// 1) esplorare possibilit� di usare la thrust per fare una sort_by_key che nel frattempo copi sull'host: obj_i_copy(k) = obj_i(front_elements(k)) for i=0, .., nobj-1 and k=0, .., size-1
	// 2) risolvere il problema dell'estrazione random nella sort di adesso (collaterale al punto 1)

	if (front_size_locked[front]==1) {
		pop->crowd_dist[front_elements_locked[0]] = INF;
		return;
	}
	if (front_size_locked[front]==2) {
		pop->crowd_dist[front_elements_locked[0]] = INF;
		pop->crowd_dist[front_elements_locked[1]] = INF;
		return;
	}
	
	for (int i=0; i<nobj; i++)
		 memcpy(tmp_locs_32s + i*size, front_elements_locked + front_offset_locked[front], front_size_locked[front]*sizeof(int));

	for (int i=0; i<nobj; i++) {
		 qsort_front_obj_gpu (tmp_locs_32s + i*size, pop->obj + i*size, 0, front_size_locked[front]-1);
		 pop->crowd_dist[tmp_locs_32s[i*size]] = INF;
	}

    for (int i=0; i<nobj; i++)
		for (int j=1; j<front_size_locked[front]-1; j++) {
			if (pop->crowd_dist[tmp_locs_32s[i*size + j]] != INF) {
				if (pop->obj[i*size + tmp_locs_32s[i*size + front_size_locked[front]-1 ]] == pop->obj[i*size + tmp_locs_32s[i*size]])
                    pop->crowd_dist[tmp_locs_32s[i*size + j]] += 0.0f;
                else
                    pop->crowd_dist[tmp_locs_32s[i*size + j]] += (pop->obj[size*i + tmp_locs_32s[i*size + j+1]] - pop->obj[size*i + tmp_locs_32s[i*size + j-1]]) / (pop->obj[size*i + tmp_locs_32s[i*size + front_size_locked[front]-1]] - pop->obj[size*i + tmp_locs_32s[i*size + 0]]);
			}
		}

	for (int j=0; j<front_size_locked[front]; j++)
		if (pop->crowd_dist[front_elements_locked[front_offset_locked[front] + j]] != INF)
			pop->crowd_dist[front_elements_locked[front_offset_locked[front] + j]] /= nobj;
    
}

void assign_rank_and_crowd_dist_gpu (population_gpu *pop_dev, population_cpu *pop, const int size) {
   
	int scale = 4;
	dim3 blocksPerGrid_dominance(size*scale, 1, 1);
	dim3 threadsPerBlock_dominance(size/scale, 1, 1);
	dim3 blocksPerGrid_front_elements(size/scale, 1, 1);
	dim3 threadsPerBlock_front_elements(scale, 1, 1);

	float alpha = 1.0f;
	float beta = 0.0f;

	NppiSize oneRowRoi;
	oneRowRoi.height = 1;
	oneRowRoi.width = size;
	int step = oneRowRoi.width*sizeof(float);

	device_ptr<int> tp_front_elements;
	device_ptr<int> tp_front_elements_end;
	device_ptr<float> tp_stencil;
	device_ptr<float> tp_idx_32f_dev = device_pointer_cast(idx_32f_dev);
	int front_size_0, front_size_1;
		
	// porta sull'host gli obj per usarli nel calcolo della crowd dist
	checkCudaErrors(hipMemcpy(pop->obj, pop_dev->obj, nobj*popsize*sizeof(float), hipMemcpyDeviceToHost));
	
	// S_dev:
	check_dominance_kernel_masked<<<blocksPerGrid_dominance, threadsPerBlock_dominance>>>(pop_dev->obj, pop_dev->constr_violation, size, nobj, S_dev, flag_dev);

	// n_dev:
	hipblasSgemv(handle, HIPBLAS_OP_N, size, size, &alpha, S_dev, size, ones_32f_dev, 1, &beta, n_dev, 1);

	// azzera crowd_dist --> sull'host (infatti usa pop e non pop_dev) <--
	memset(pop->crowd_dist, 0, size*sizeof(float));
	
	///////// PRIMO FRONTE ///////////

	pop->front_counter = 0;
	front_offset_locked[0] = 0;
	
	// 1) vettore binario degli elementi:
	nppiThreshold_LTValGTVal_32f_C1R(n_dev, step, tmp_locs_32f_dev + size, step, oneRowRoi, 1, 1, 0, 0);

	// 3) trova indici e determina numero elementi:
	tp_front_elements = device_pointer_cast(front_elements_dev);
	tp_stencil = device_pointer_cast(tmp_locs_32f_dev + size);
	
	tp_front_elements_end = copy_if(tp_idx_32f_dev, tp_idx_32f_dev + size, tp_stencil, tp_front_elements, is_one<int>());
	front_size_0 = tp_front_elements_end - tp_front_elements;
	front_size_locked[0] = front_size_0;

	// 4) assegna crowd_dist --> sull'host (infatti usa pop e non pop_dev) <--
	assign_crowd_dist_gpu (pop, size, pop->front_counter);

	///////// SECONDO FRONTE /////////

	pop->front_counter++;
	front_offset_locked[1] = front_size_0;
	
	// 1) vettore binario degli elementi:
	hipblasSgemv(handle, HIPBLAS_OP_N, size, size, &alpha, S_dev, size, tmp_locs_32f_dev + size, 1, &beta, tmp_locs_32f_dev, 1);
	nppsSub_32f_I(tmp_locs_32f_dev, n_dev, size);
	nppiThreshold_LTValGTVal_32f_C1R(n_dev, step, tmp_locs_32f_dev, step, oneRowRoi, 1, 1, 0, 0);
	nppsSub_32f_I(tmp_locs_32f_dev + size, tmp_locs_32f_dev, size);
	
	// 3) trova indici e determina numero elementi:
	tp_front_elements = device_pointer_cast(front_elements_dev + front_size_0);
	tp_stencil = device_pointer_cast(tmp_locs_32f_dev);
		
	tp_front_elements_end = copy_if(tp_idx_32f_dev, tp_idx_32f_dev + size, tp_stencil, tp_front_elements, is_one<int>());
	front_size_1 = tp_front_elements_end - tp_front_elements;
	front_size_locked[1] = front_size_1;

	///////// FRONTI SUCCESSIVI /////////
	
	while (front_size_locked[pop->front_counter]>0) {

		pop->front_counter++;
		front_size_locked[pop->front_counter] = 0;
		front_offset_locked[pop->front_counter] = front_offset_locked[pop->front_counter-1] + front_size_locked[pop->front_counter-1];

		calc_next_front<<<blocksPerGrid_front_elements,threadsPerBlock_front_elements>>>(n_dev, S_dev, size, 
						front_elements_dev, front_size_dev, front_offset_dev, pop->front_counter);

		assign_crowd_dist_gpu (pop, size, pop->front_counter-1);

		hipDeviceSynchronize();
	}

	// copia sul device la crowd_dist calcolata
	checkCudaErrors(hipMemcpy(pop_dev->crowd_dist, pop->crowd_dist, popsize*sizeof(float), hipMemcpyHostToDevice));
}    
