#include "hip/hip_runtime.h"
#include <thrust/iterator/counting_iterator.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/sort.h>

#include "global_cuda.cuh"

using namespace thrust;
using namespace thrust::placeholders;

__constant__ float *min_realvar_dev;
__constant__ float *max_realvar_dev;

__constant__ float pcross_real_dev;
__constant__ int nrealcross_dev;
__constant__  float eta_c_dev;

__constant__ float pmut_real_dev;
__constant__ int nrealmut_dev;
__constant__ float eta_m_dev;

void copy_constants() {
	
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(min_realvar_dev), &min_realvar, nreal*sizeof(float)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(max_realvar_dev), &max_realvar, nreal*sizeof(float)));
	
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(&pcross_real_dev), &pcross_real, sizeof(float)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(&nrealcross_dev), &nrealcross, sizeof(int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(&eta_c_dev), &eta_c, sizeof(float)));
	
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(&pmut_real_dev), &pmut_real, sizeof(float)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(&nrealmut_dev), &nrealmut, sizeof(int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(&eta_m_dev), &eta_m, sizeof(float)));
}

void selection_gpu (population_gpu *old_pop, population_gpu *new_pop, int size) {
    
	float *parent12 = tmp_locs_32f_dev;
	
	float *p12 = tmp_locs_32f_dev + 2*size;
	
	float *rand = tmp_locs_32f_dev + 3*size; // per la dimensione di rand1 vedi sotto
	  
	dim3 threadsPerBlock_rndvec(DIM,1,1);
	dim3 blocksPerGrid_rndvec(1,1,1);

	dim3 threadsPerBlock_tournament(size,1,1);
	dim3 blocksPerGrid_tournament(1,1,1);

	dim3 threadsPerBlock_realcross(size,1,1);
	dim3 blocksPerGrid_realcross(nreal,1,1);

	extract_rndvec_kernel<<<blocksPerGrid_rndvec, threadsPerBlock_rndvec>>>(rand, 2*size);

	checkCudaErrors(hipMemcpy(parent12, idx_32f_dev, size*sizeof(float), hipMemcpyDeviceToDevice));
	checkCudaErrors(hipMemcpy(parent12 + size, idx_32f_dev, size*sizeof(float), hipMemcpyDeviceToDevice));
	
	sort_by_key(rand, rand + size, parent12);
	sort_by_key(rand + size, rand + 2*size, parent12 + size);

	// serbatoio di numeri random 
	extract_rndvec_kernel<<<blocksPerGrid_rndvec, threadsPerBlock_rndvec>>>(rand, size + // tournament
																	 size/2 + // tmp1 soglia pcross_real
																	 3*nreal*size/2); // tmp2a, tmp2b, tmp2c
	
	// mettere due stream e lanciare le due coppie di kernel in concorrenza
	tournament_kernel<<<blocksPerGrid_tournament,threadsPerBlock_tournament>>>(parent12, p12, old_pop->crowd_dist, flag_dev, size, rand);
	
	// 3) nreal_cross � la somma degli elementi pari a uno dentro a ...

	// 3) soglia 0.5 sui numeri rand1(size + size/2, size + size/2 + 2*nreal*size/2) e mette il risultato in tmp2
	// 4) il primo kernel usera tmp2(0, 2*nreal*size/4) e il secondo tmp2(2*nreal*size/4, 2*nreal*size/2)
		
	realcross_kernel<<<blocksPerGrid_realcross,threadsPerBlock_realcross>>>(old_pop->xreal, new_pop->xreal, size,
																				p12,
																				rand + size,
																				rand + size + size/2,
																				rand + size + size/2 + nreal*size/2,
																				rand + size + size/2 + 2*nreal*size/2);
}

__global__ void tournament_kernel (float* parent, float* p, float* crowd_dist_loc, char* flag, const int size, float *rand_vec) {

	unsigned tid = threadIdx.x + blockIdx.x*blockDim.x; // da 0 a size-1
	unsigned idx = tid + (tid%2) + (size-2)*((tid%4)>1);
	int ind1 = (int)parent[idx];
	int ind2 = (int)parent[idx + 1];

	float crowd_dist_loc1 = crowd_dist_loc[ind1];
	float crowd_dist_loc2 = crowd_dist_loc[ind2];

	int flag_loc = (int)flag[ind2*size + ind1];

	float rand_num = rand_vec[tid];

	p[tid] = (flag_loc==-1)*ind2 + 
		      (flag_loc==1)*ind1 + 
		      (flag_loc==0)*( (crowd_dist_loc1>crowd_dist_loc2)*ind1 + 
			                (crowd_dist_loc2>crowd_dist_loc1)*ind2 + 
							(crowd_dist_loc1==crowd_dist_loc2)*( (rand_num<=0.5f)*ind1 + (rand_num>0.5f)*ind2 ) );
}

__global__ void realcross_kernel (float* parent_x, float* child_x, int size, float* p, float *tmp1, float *tmp2a, float *tmp2b, float *tmp2c) {

	unsigned tid = threadIdx.x + blockIdx.x*blockDim.x; // da 0 a nreal*size-1
	unsigned c = tid/size;
	unsigned offc = c*size;
	unsigned pidx = tid - offc; // da 0 a size-1
	unsigned rem = (pidx%2);
	pidx -= rem; 
	int p1 = (int)p[pidx];
	int p2 = (int)p[pidx + 1];
	
	float xp1 = parent_x[offc + p1];
	float xp2 = parent_x[offc + p2];

	// 1) ogni thread soglia pcross_real su tmp1(0,size/2)
	bool thresh_ind = tmp1[pidx/2]<=pcross_real_dev;
	// 2) ogni thread soglia 0.5 su tmp2(0, nreal*size/2) fa la and e la mette in thresh_x1
	bool thresh_x1 = thresh_ind && ( tmp2a[tid/2]<=0.5f );
	// 3) ogni thread soglia EPS su fabs(xp1-xp2) fa la and e la mette in thresh_x2
	bool thresh_x2 = thresh_x1 && (fabs(xp1-xp2)>EPS);
	
	float y1 = min(xp1, xp2);
	float y2 = max(xp1, xp2);
	
	float yl = min_realvar_dev[c];
	float yu = max_realvar_dev[c];

	float beta1 = 1.0f + (2.0f*fabs(y1-yl)/(y2-y1));
	float beta2 = 1.0f + (2.0f*fabs(y2-yu)/(y2-y1));

	float alpha1 = 2.0f - powf(beta1,-(eta_c_dev+1.0f));
	float alpha2 = 2.0f - powf(beta2,-(eta_c_dev+1.0f));

	float rand = tmp2b[tid/2];
	bool thresh_rand1 = (rand <= (1.0f/alpha1));
	bool thresh_rand2 = (rand <= (1.0f/alpha2));

	float betaq1 = powf (rand*alpha1, 1.0f/(eta_c_dev+1.0f))*thresh_rand1 + powf (1.0f/(2.0f - rand*alpha1), 1.0f/(eta_c_dev+1.0f))*(1-thresh_rand1);
	float betaq2 = powf (rand*alpha2, 1.0f/(eta_c_dev+1.0f))*thresh_rand2 + powf (1.0f/(2.0f - rand*alpha2), 1.0f/(eta_c_dev+1.0f))*(1-thresh_rand2);
					
	float c1 = 0.5f * ((y1+y2) - betaq1*(y2-y1));
	float c2 = 0.5f * ((y1+y2) + betaq2*(y2-y1));

    c1 = (c1<yl)? yl : c1; 
	c1 = (c1>yu)? yu : c1; 
	c2 = (c2<yl)? yl : c2; 
	c2 = (c2>yu)? yu : c2; 

	rand = tmp2c[tid/2];
	bool thresh_rand = (rand<=0.5f);
	
	// 4a) adesso i punti per cui thresh_x2 vale 1 sono da modificare nel modo seguente:
	float child_x_changed = c1 * ( (1-rem)*(1-thresh_rand) + rem*thresh_rand ) + c2 * ( (1-rem)*thresh_rand + rem*(1-thresh_rand) );
	
	// 4b) gli altri nel modo seguente:
	float child_x_unchanged = xp1*(1-rem) + xp2*rem;
   
	child_x[tid] = child_x_changed * thresh_x2 + child_x_unchanged * (1-thresh_x2);
}