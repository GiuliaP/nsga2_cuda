#include "hip/hip_runtime.h"
#include "global_cuda.cuh"

void mutation_gpu (population_gpu *pop, int size) {

	if (nreal!=0)
		for (int i=0; i<size; i++)
			realmutate_gpu (pop, size, i);
}

void realmutate_gpu (population_gpu *pop, int size, int ind) {
   
    float rnd;
	float delta1, delta2;
	float mut_pow, deltaq;
    float y, yl, yu;
	float val, xy;
    
	for (int j=0; j<nreal; j++)
        if (randomperc() <= pmut_real) {
            y = pop->xreal[j*size + ind];
            yl = min_realvar[j];
            yu = max_realvar[j];
            delta1 = (y-yl)/(yu-yl);
            delta2 = (yu-y)/(yu-yl);
            rnd = randomperc();
            mut_pow = 1.0f/(eta_m+1.0f);
            if (rnd <= 0.5f) {
                xy = 1.0f-delta1;
                val = 2.0f*rnd + (1.0f-2.0f*rnd)*pow(xy,eta_m+1.0f);
                deltaq =  pow(val,mut_pow) - 1.0f;
            }
            else {
                xy = 1.0f-delta2;
                val = 2.0f*(1.0f-rnd)+2.0f*(rnd-0.5f)*pow(xy,eta_m+1.0f);
                deltaq = 1.0f - (pow(val,mut_pow));
            }
            y = y + deltaq*(yu-yl);
            if (y<yl)
                y = yl;
            if (y>yu)
                y = yu;
            pop->xreal[j*size + ind] = y;
            nrealmut+=1;
        }
}