#include "global_cuda.cuh"

void allocate_pop_gpu (population_gpu *pop, int size) {

	if (nreal != 0)
       checkCudaErrors(hipMalloc((void**)&(pop->xreal), size*nreal*sizeof(float)));
 
    checkCudaErrors(hipMalloc((void**)&(pop->obj), size*nobj*sizeof(float)));
    
	if (ncon != 0)
        checkCudaErrors(hipMalloc((void**)&(pop->constr), size*ncon*sizeof(float)));

	checkCudaErrors(hipMalloc((void**)&(pop->crowd_dist), size*sizeof(float)));
    checkCudaErrors(hipMalloc((void**)&(pop->constr_violation), size*sizeof(float)));


}
void free_pop_gpu (population_gpu *pop) {

	if (nreal != 0)
       checkCudaErrors(hipFree(pop->xreal));
 
    checkCudaErrors(hipFree(pop->obj));
    
	if (ncon != 0)
        checkCudaErrors(hipFree(pop->constr));

	checkCudaErrors(hipFree(pop->crowd_dist));
    checkCudaErrors(hipFree(pop->constr_violation));
}

void upload_pop (population_gpu *pop_dev, population_cpu *pop, int size) {

	if (nreal != 0)
       checkCudaErrors(hipMemcpy((void*)pop_dev->xreal, (void*)pop->xreal, size*nreal*sizeof(float), hipMemcpyHostToDevice));
 
    checkCudaErrors(hipMemcpy((void*)pop_dev->obj, (void*)pop->obj, size*nobj*sizeof(float), hipMemcpyHostToDevice));
    
	if (ncon != 0)
        checkCudaErrors(hipMemcpy((void*)pop_dev->constr, pop->constr, size*ncon*sizeof(float), hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy((void*)pop_dev->crowd_dist, (void*)pop->crowd_dist, size*sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy((void*)pop_dev->constr_violation, (void*)pop->constr_violation, size*sizeof(float), hipMemcpyHostToDevice));
}
void download_pop (population_cpu *pop, population_gpu *pop_dev, int size) {

	if (nreal != 0)
       checkCudaErrors(hipMemcpy((void*)pop->xreal, (void*)pop_dev->xreal, size*nreal*sizeof(float), hipMemcpyDeviceToHost));
 
    checkCudaErrors(hipMemcpy((void*)pop->obj, (void*)pop_dev->obj, size*nobj*sizeof(float), hipMemcpyDeviceToHost));
    
	if (ncon != 0)
        checkCudaErrors(hipMemcpy((void*)pop->constr, pop_dev->constr, size*ncon*sizeof(float), hipMemcpyDeviceToHost));

	checkCudaErrors(hipMemcpy((void*)pop->crowd_dist, (void*)pop_dev->crowd_dist, size*sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy((void*)pop->constr_violation, (void*)pop_dev->constr_violation, size*sizeof(float), hipMemcpyDeviceToHost));
}