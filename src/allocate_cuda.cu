#include "global_cuda.cuh"

void allocate_pop_gpu (population *pop, int size) {

	if (nreal != 0)
       checkCudaErrors(hipMalloc((void**)&(pop->xreal), size*nreal*sizeof(float)));
 
    checkCudaErrors(hipMalloc((void**)&(pop->obj), size*nobj*sizeof(float)));
    
	if (ncon != 0)
        checkCudaErrors(hipMalloc((void**)&(pop->constr), size*ncon*sizeof(float)));

	checkCudaErrors(hipMalloc((void**)&(pop->rank), size*sizeof(int))); 
	checkCudaErrors(hipMalloc((void**)&(pop->crowd_dist), size*sizeof(float)));
    checkCudaErrors(hipMalloc((void**)&(pop->constr_violation), size*sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&(pop->front_size), size*sizeof(int)));
	checkCudaErrors(hipMalloc((void**)&(pop->front_offset), size*sizeof(int)));
	checkCudaErrors(hipMalloc((void**)&(pop->front_elements), size*sizeof(int)));
}
void free_pop_gpu (population *pop) {

	if (nreal != 0)
       checkCudaErrors(hipFree(pop->xreal));
 
    checkCudaErrors(hipFree(pop->obj));
    
	if (ncon != 0)
        checkCudaErrors(hipFree(pop->constr));

	checkCudaErrors(hipFree(pop->rank)); 
	checkCudaErrors(hipFree(pop->crowd_dist));
    checkCudaErrors(hipFree(pop->constr_violation));
	checkCudaErrors(hipFree(pop->front_size));
	checkCudaErrors(hipFree(pop->front_offset));
	checkCudaErrors(hipFree(pop->front_elements));
}

void upload_pop (population *pop, population *pop_host, int size) {

	if (nreal != 0)
       checkCudaErrors(hipMemcpy((void*)pop->xreal, (void*)pop_host->xreal, size*nreal*sizeof(float), hipMemcpyHostToDevice));
 
    checkCudaErrors(hipMemcpy((void*)pop->obj, (void*)pop_host->obj, size*nobj*sizeof(float), hipMemcpyHostToDevice));
    
	if (ncon != 0)
        checkCudaErrors(hipMemcpy((void*)pop->constr, pop_host->constr, size*ncon*sizeof(float), hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy((void*)pop->rank, (void*)pop_host->rank, size*sizeof(int), hipMemcpyHostToDevice)); 
	checkCudaErrors(hipMemcpy((void*)pop->crowd_dist, (void*)pop_host->crowd_dist, size*sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy((void*)pop->constr_violation, (void*)pop_host->constr_violation, size*sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy((void*)pop->front_size, (void*)pop_host->front_size, size*sizeof(int), hipMemcpyHostToDevice)); 
	checkCudaErrors(hipMemcpy((void*)pop->front_offset, (void*)pop_host->front_offset, size*sizeof(int), hipMemcpyHostToDevice)); 
	checkCudaErrors(hipMemcpy((void*)pop->front_elements, (void*)pop_host->front_elements, size*sizeof(int), hipMemcpyHostToDevice)); 
}
void download_pop (population *pop_host, population *pop, int size) {

	if (nreal != 0)
       checkCudaErrors(hipMemcpy((void*)pop_host->xreal, (void*)pop->xreal, size*nreal*sizeof(float), hipMemcpyDeviceToHost));
 
    checkCudaErrors(hipMemcpy((void*)pop_host->obj, (void*)pop->obj, size*nobj*sizeof(float), hipMemcpyDeviceToHost));
    
	if (ncon != 0)
        checkCudaErrors(hipMemcpy((void*)pop_host->constr, pop->constr, size*ncon*sizeof(float), hipMemcpyDeviceToHost));

	checkCudaErrors(hipMemcpy((void*)pop_host->rank, (void*)pop->rank, size*sizeof(int), hipMemcpyDeviceToHost)); 
	checkCudaErrors(hipMemcpy((void*)pop_host->crowd_dist, (void*)pop->crowd_dist, size*sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy((void*)pop_host->constr_violation, (void*)pop->constr_violation, size*sizeof(float), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy((void*)pop_host->front_size, (void*)pop->front_size, size*sizeof(int), hipMemcpyDeviceToHost)); 
	checkCudaErrors(hipMemcpy((void*)pop_host->front_offset, (void*)pop->front_offset, size*sizeof(int), hipMemcpyDeviceToHost)); 
	checkCudaErrors(hipMemcpy((void*)pop_host->front_elements, (void*)pop->front_elements, size*sizeof(int), hipMemcpyDeviceToHost)); 
}